#include "hip/hip_runtime.h"
// Approximation of Pi using a simple, and not optimized, CUDA program
// Copyleft Alessandro Re

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

typedef unsigned long long Count;

const Count WARP_SIZE = 32; // Warp size
const Count NBLOCKS = 64; // Number of total cuda cores on my GPU

__global__ void picount(Count *totals, Count iterPerThread) {
	__shared__ Count counter[WARP_SIZE];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	hiprandState_t rng;
	hiprand_init(clock64(), tid, 0, &rng);

  Count cnt = 0;
	// Computation loop
	for (int i = 0; i < iterPerThread; i++) {
		float x = hiprand_uniform(&rng); 
    float y = hiprand_uniform(&rng);
    if(x*x + y*y <= 1.0) cnt++;
  }
  
  counter[threadIdx.x] = cnt;

	if (threadIdx.x == 0) {
		totals[blockIdx.x] = 0;
		for (int i = 0; i < WARP_SIZE; i++) {
			totals[blockIdx.x] += counter[i];
		}
	}
}

int main(int argc, char **argv) {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	if (nDevices < 1) {
		printf("CUDA device missing! Do you need to use optirun?\n");
		return 1;
	}

	printf("------CUDA Devices------\n");
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
  }
	
	
  Count totalIterations;
  scanf("%lld", &totalIterations);

  Count totalThreads = (WARP_SIZE * NBLOCKS);
  Count iterPerThread = (totalIterations + totalThreads - 1LL)/(totalThreads);
  totalIterations = iterPerThread * totalThreads;
  printf("\nStarting simulation with %lld blocks, %lld threads per block (warps), and a total of %lld iterations\n", NBLOCKS, WARP_SIZE, totalIterations);

  Count *hOut, *dOut;
	hOut = new Count[NBLOCKS]; 
  hipMalloc(&dOut, sizeof(Count) * NBLOCKS);

	picount<<<NBLOCKS, WARP_SIZE>>>(dOut, iterPerThread);

	hipMemcpy(hOut, dOut, sizeof(Count) * NBLOCKS, hipMemcpyDeviceToHost);
	hipFree(dOut);

	Count total = 0;
	for (int i = 0; i < NBLOCKS; i++) {
		total += hOut[i];
	}
	printf("Approximated PI using %lld random tests\n", totalIterations);

	double pi = 4.0 * (double)total/(double)totalIterations;
	printf("PI ~= %.9lf\n", pi);

	return 0;
}